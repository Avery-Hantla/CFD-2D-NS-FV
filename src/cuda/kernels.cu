#ifndef KERNELS_CU
    #define KERNELS_CU

    #include <hip/hip_runtime.h>
    #include <iostream>

    __global__ void d_rusanov(int N, double d_cl_Vn[], double d_c2_Vn[], double d_c1_c[], double d_c2_c[], double d_Vn_avg[], double d_c_avg[], double d_Q_c1_p1[], double d_Q_c1_p2[], double d_Q_c1_p3[], double d_Q_c1_p4[], double d_Q_c2_p1[], double d_Q_c2_p2[], double d_Q_c2_p3[], double d_Q_c2_p4[], double d_F_c1_p1[], double d_F_c1_p2[], double d_F_c1_p3[], double d_F_c1_p4[], double d_F_c2_p1[], double d_F_c2_p2[], double d_F_c2_p3[], double d_F_c2_p4[], double d_F_rusanov_p1[], double d_F_rusanov_p2[], double d_F_rusanov_p3[], double d_F_rusanov_p4[]) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < N) {
            // Find Vn and c averag 
            double Vn_avg = (d_cl_Vn[idx] + d_c2_Vn[idx])/2;
            double c_avg = (d_c1_c[idx] + d_c2_c[idx])/2;

            // Save for time step calculation
            d_Vn_avg[idx] = Vn_avg;
            d_c_avg[idx] = c_avg;

            // Compute Rusanov Flux
            d_F_rusanov_p1[idx] = (d_F_c1_p1[idx]+d_F_c2_p1[idx])/2 - (std::abs(Vn_avg) + c_avg)/2 * (d_Q_c2_p1[idx] - d_Q_c1_p1[idx]);
            d_F_rusanov_p2[idx] = (d_F_c1_p2[idx]+d_F_c2_p2[idx])/2 - (std::abs(Vn_avg) + c_avg)/2 * (d_Q_c2_p2[idx] - d_Q_c1_p2[idx]);
            d_F_rusanov_p3[idx] = (d_F_c1_p3[idx]+d_F_c2_p3[idx])/2 - (std::abs(Vn_avg) + c_avg)/2 * (d_Q_c2_p3[idx] - d_Q_c1_p3[idx]);
            d_F_rusanov_p4[idx] = (d_F_c1_p4[idx]+d_F_c2_p4[idx])/2 - (std::abs(Vn_avg) + c_avg)/2 * (d_Q_c2_p4[idx] - d_Q_c1_p4[idx]);

            printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
        }
    }

#endif