#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////
//           Function To Compute Rusanov Flux
/////////////////////////////////////////////////////////

#include "../support/class_q.hpp"
#include "../support/class_mesh.hpp"
#include "../support/class_f.hpp"
#include "../support/class_flow.hpp"
#include "../support/struct_size.hpp"
#include "../support/struct_BC.hpp"

// #include "../solver/rusanov.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include "cuda.cuh"

// temp move to function in main.cpp
int maxThreadsperblock() {
    // current CUDA device ID
    int device;

    // object to store properties of the CUDA device
    hipDeviceProp_t props;

    // ID of the currently active CUDA device
    hipGetDevice(&device);

    // Retrieve properties and store in 'props'
    hipGetDeviceProperties(&props, device);

    // Return max number of threads per block
    return props.maxThreadsPerBlock;
}

__global__ void d_rusanov(int N, double d_cl_Vn[], double d_c2_Vn[], double d_c1_c[], double d_c2_c[], double d_Vn_avg[], double d_c_avg[], double d_Q_c1_p1[], double d_Q_c1_p2[], double d_Q_c1_p3[], double d_Q_c1_p4[], double d_Q_c2_p1[], double d_Q_c2_p2[], double d_Q_c2_p3[], double d_Q_c2_p4[], double d_F_c1_p1[], double d_F_c1_p2[], double d_F_c1_p3[], double d_F_c1_p4[], double d_F_c2_p1[], double d_F_c2_p2[], double d_F_c2_p3[], double d_F_c2_p4[], double d_F_rusanov_p1[], double d_F_rusanov_p2[], double d_F_rusanov_p3[], double d_F_rusanov_p4[]) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        // Find Vn and c averag 
        double Vn_avg = (d_cl_Vn[idx] + d_c2_Vn[idx])/2;
        double c_avg = (d_c1_c[idx] + d_c2_c[idx])/2;

        // Save for time step calculation
        d_Vn_avg[idx] = Vn_avg;
        d_c_avg[idx] = c_avg;

        // Compute Rusanov Flux
        d_F_rusanov_p1[idx] = (d_F_c1_p1[idx]+d_F_c2_p1[idx])/2 - (std::abs(Vn_avg) + c_avg)/2 * (d_Q_c2_p1[idx] - d_Q_c1_p1[idx]);
        d_F_rusanov_p2[idx] = (d_F_c1_p2[idx]+d_F_c2_p2[idx])/2 - (std::abs(Vn_avg) + c_avg)/2 * (d_Q_c2_p2[idx] - d_Q_c1_p2[idx]);
        d_F_rusanov_p3[idx] = (d_F_c1_p3[idx]+d_F_c2_p3[idx])/2 - (std::abs(Vn_avg) + c_avg)/2 * (d_Q_c2_p3[idx] - d_Q_c1_p3[idx]);
        d_F_rusanov_p4[idx] = (d_F_c1_p4[idx]+d_F_c2_p4[idx])/2 - (std::abs(Vn_avg) + c_avg)/2 * (d_Q_c2_p4[idx] - d_Q_c1_p4[idx]);

        // printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
    }
}

void rusanov(class_F* F_rusanov, class_Q* Qface_c1, class_Q* Qface_c2, class_mesh* mesh, class_flow* freestream, struct_size* size, struct_BC* BC) {
    class_F F_c1, F_c2;
    F_c1.init(size->num_faces, freestream->gamma);
    F_c2.init(size->num_faces, freestream->gamma);
    
    F_c1.update(mesh, Qface_c1, size); 
    F_c2.update(mesh, Qface_c2, size);

    // Device pointers for vectors
    double *d_cl_Vn, *d_c2_Vn, *d_c1_c, *d_c2_c;
    double *d_Vn_avg, *d_c_avg;

    double *d_Q_c1_p1, *d_Q_c1_p2, *d_Q_c1_p3, *d_Q_c1_p4; 
    double *d_Q_c2_p1, *d_Q_c2_p2, *d_Q_c2_p3, *d_Q_c2_p4;

    double *d_F_c1_p1, *d_F_c1_p2, *d_F_c1_p3, *d_F_c1_p4;
    double *d_F_c2_p1, *d_F_c2_p2, *d_F_c2_p3, *d_F_c2_p4;

    double *d_F_rusanov_p1, *d_F_rusanov_p2, *d_F_rusanov_p3, *d_F_rusanov_p4;

    // Allocate memory on the device for vectors
    hipMalloc((void **)&d_cl_Vn, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_c2_Vn, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_c1_c, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_c2_c, size->num_faces * sizeof(double));

    hipMalloc((void **)&d_Vn_avg, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_c_avg, size->num_faces * sizeof(double));
    
    hipMalloc((void **)&d_Q_c1_p1, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_Q_c1_p2, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_Q_c1_p3, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_Q_c1_p4, size->num_faces * sizeof(double));

    hipMalloc((void **)&d_Q_c2_p1, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_Q_c2_p2, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_Q_c2_p3, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_Q_c2_p4, size->num_faces * sizeof(double));

    hipMalloc((void **)&d_F_c1_p1, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_F_c1_p2, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_F_c1_p3, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_F_c1_p4, size->num_faces * sizeof(double));

    hipMalloc((void **)&d_F_c2_p1, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_F_c2_p2, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_F_c2_p3, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_F_c2_p4, size->num_faces * sizeof(double));

    hipMalloc((void **)&d_F_rusanov_p1, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_F_rusanov_p2, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_F_rusanov_p3, size->num_faces * sizeof(double));
    hipMalloc((void **)&d_F_rusanov_p4, size->num_faces * sizeof(double));

    // Copy vectors from host to device
    hipMemcpy(d_cl_Vn, &F_c1.Vn[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_c2_Vn, &F_c2.Vn[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_c1_c, &Qface_c1->c[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_c2_c, &Qface_c2->c[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_Q_c1_p1, &Qface_c1->p1[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Q_c1_p2, &Qface_c1->p2[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Q_c1_p3, &Qface_c1->p3[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Q_c1_p4, &Qface_c1->p4[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_Q_c2_p1, &Qface_c2->p1[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Q_c2_p2, &Qface_c2->p2[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Q_c2_p3, &Qface_c2->p3[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Q_c2_p4, &Qface_c2->p4[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_F_c1_p1, &F_c1.p1[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_F_c1_p2, &F_c1.p2[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_F_c1_p3, &F_c1.p3[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_F_c1_p4, &F_c1.p4[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_F_c2_p1, &F_c2.p1[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_F_c2_p2, &F_c2.p2[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_F_c2_p3, &F_c2.p3[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_F_c2_p4, &F_c2.p4[0], size->num_faces * sizeof(double), hipMemcpyHostToDevice);

    int nThreads = maxThreadsperblock();
    int nBlocks = ceil((double)size->num_faces / nThreads);

    // Run Kernel
    d_rusanov <<<nBlocks, nThreads>>> (size->num_faces, d_cl_Vn, d_c2_Vn, d_c1_c, d_c2_c, d_Vn_avg, d_c_avg, d_Q_c1_p1, d_Q_c1_p2, d_Q_c1_p3, d_Q_c1_p4, d_Q_c2_p1, d_Q_c2_p2, d_Q_c2_p3, d_Q_c2_p4, d_F_c1_p1, d_F_c1_p2, d_F_c1_p3, d_F_c1_p4, d_F_c2_p1, d_F_c2_p2, d_F_c2_p3, d_F_c2_p4, d_F_rusanov_p1, d_F_rusanov_p2, d_F_rusanov_p3, d_F_rusanov_p4);

    // Check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();

    // Copy vectors from device to host
    hipMemcpy(&F_rusanov->p1[0], d_F_rusanov_p1, size->num_faces * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&F_rusanov->p2[0], d_F_rusanov_p2, size->num_faces * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&F_rusanov->p3[0], d_F_rusanov_p3, size->num_faces * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&F_rusanov->p4[0], d_F_rusanov_p4, size->num_faces * sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(&Qface_c1->Vn_avg[0], d_Vn_avg, size->num_faces * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&Qface_c2->Vn_avg[0], d_Vn_avg, size->num_faces * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&Qface_c1->c_avg[0], d_c_avg, size->num_faces * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&Qface_c2->c_avg[0], d_c_avg, size->num_faces * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipDeviceReset();
}
