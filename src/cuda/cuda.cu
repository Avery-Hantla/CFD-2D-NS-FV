#ifndef CUDA_HPP
    #define CUDA_HPP
    
    #include <hip/hip_runtime.h>

    // #include "cuda/kernels.cu"
    // __global__ void d_rusanov(int N, double d_cl_Vn[], double d_c2_Vn[], double d_c1_c[], double d_c2_c[], double d_Vn_avg[], double d_c_avg[], double d_Q_c1_p1[], double d_Q_c1_p2[], double d_Q_c1_p3[], double d_Q_c1_p4[], double d_Q_c2_p1[], double d_Q_c2_p2[], double d_Q_c2_p3[], double d_Q_c2_p4[], double d_F_c1_p1[], double d_F_c1_p2[], double d_F_c1_p3[], double d_F_c1_p4[], double d_F_c2_p1[], double d_F_c2_p2[], double d_F_c2_p3[], double d_F_c2_p4[], double d_F_rusanov_p1[], double d_F_rusanov_p2[], double d_F_rusanov_p3[], double d_F_rusanov_p4[]);

    int maxThreadsperblock() {
        // current CUDA device ID
        int device;

        // object to store properties of the CUDA device
        hipDeviceProp_t props;

        // ID of the currently active CUDA device
        hipGetDevice(&device);

        // Retrieve properties and store in 'props'
        hipGetDeviceProperties(&props, device);

        // Return max number of threads per block
        return props.maxThreadsPerBlock;
    }
#endif